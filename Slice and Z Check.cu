//TODO: TEST OUT IF THE SLICE AND Z FOR THE DIAGONAL IMPLEMENTATION WORKS AS INTENDED
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void insertValues(int *arr, int slice, int z, int rowLength, int colLength){
    int id = threadIdx.x + blockIdx.x*blockDim.x;
    //row gets shifted down depending on the value of z
    int row = id + z;

    int col;
    //col starts with slice until its higher than column length
    if(slice < colLength){
        col = slice - id;
    }
    else{
        col = (colLength - 1) - id;
    }

    //the position in 1D array is calculated with (row x column length) + column
    int pos = (row * colLength) + col;
    arr[(row * colLength) + col] = slice;
    printf("pos: (%d,%u) \n %i  \n", row, col, arr[(row*colLength) + col]);
    //printf("%i  \n", arr[(row*rowLength) + col]);
}

__global__ void printArr(int *arr, int rowLength, int colLength){
    int id = threadIdx.x + blockIdx.x*blockDim.x;

    printf("Array from kernel:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            int pos = (i*colLength)+j;
            printf("%d:%i  ", pos, arr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");
}


int main(int argc, char *argv[]){
    //testing 2D arrays with 1D array representation too
    int *arr, *hostArr;

    int rowLength = 4;
    int colLength = 5;

    int size = rowLength*colLength;

    //allocate memory
    hostArr = (int*) std::malloc(size*sizeof(int));
    hipMalloc(&arr, size*sizeof(int));


    //print before running the kernel(s)
    printf("Array before:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            printf("%i  ", hostArr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");

    //just as a note:
    //the for loop below for the diagonal implementation uses slice as the baseline
    //slice functions as to determine the position of the diagonal for the iteration
    //example:
    //lets say slice = 1, visually on a 3x3 array its like this:
    //| 0  s  0 |
    //| s  0  0 |  <-- with s being the representation of the slice
    //| 0  0  0 |
    //
    //this continues until slice is higher the length of the column,
    //where the diagonal cannot be calculated with only the column as reference
    //the diagonal needs to continue through the 'bottom' half of the array 
    //
    //this is where z starts to function
    //after the slice is higher than the column, z is calculated using this formula:
    //z = slice - column + 1
    //lets say the slice is only 1 higher than the column,
    //this means the z is 1, and with that, the diagonal "shifts down" by 1
    //an example on a 3x3 array with z = 1:
    //| 0  0  0 |
    //| 0  0  s |
    //| 0  s  0 |

    int z, tSize;
    int bSize = 1;
    
	for(int slice=0; slice < colLength*2; slice++){
		if(slice < colLength){
			z = 0;
            tSize = slice + 1;
		}
		else{
			z = slice - colLength + 1;
            tSize = (colLength + 1) - z - 1;
		}
		
        //calculate thread and blocks used
		if(tSize <= 256){
			bSize = 1;
		}
		else{
			bSize = tSize / 256;
            tSize = 256;
		}

		insertValues<<<bSize, tSize>>>(arr, slice, z, rowLength, colLength);
	}
    printf("\n");

    //copy device array from insertValues to host array
    hipMemcpy(hostArr, arr, size*sizeof(int), hipMemcpyDeviceToHost);

    //printArr<<<1,1>>>(arr, rowLength, colLength);
    
    //print after running the kernel(s)
    printf("Array after:\n");
    for(int i= 0; i<rowLength; i++){
        for(int j= 0; j<colLength; j++){
            //printf("%d,%u :%i  ", i, j, hostArr[(i*colLength) + j]);
            printf("%i  ", hostArr[(i*colLength) + j]);
        }
        printf("\n");
    }
    printf("\n");
}
